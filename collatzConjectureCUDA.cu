#include <cstdio>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <sys/time.h>

static const int ThreadsPerBlock = 512;

static __global__ void collatz(const long upper, int* const maxlen)
{
  const long i = threadIdx.x + blockIdx.x * (long)blockDim.x;
  // compute sequence lengths
  if (i < (upper + 1)/2) {
    long val = 2*i + 1; // translate to i-th odd
    int len = 1;
    while (val != 1) {
      len++;
      if ((val % 2) == 0) {
        val = val / 2;  // even
      } else {
        val = 3 * val + 1;  // odd
      }
    }
    if(len > *maxlen) atomicMax(maxlen, len); // per instructions
  }
}

static void CheckCuda()
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d: %s\n", e, hipGetErrorString(e));
    exit(-1);
  }
}

int main(int argc, char *argv[])
{
  printf("Collatz v1.2\n");

  // check command line
  if (argc != 2) {fprintf(stderr, "USAGE: %s upper_bound\n", argv[0]); exit(-1);}
  const long upper = atol(argv[1]);
  if (upper < 5) {fprintf(stderr, "ERROR: upper_bound must be at least 5\n"); exit(-1);}
  if ((upper % 2) != 1) {fprintf(stderr, "ERROR: upper_bound must be an odd number\n"); exit(-1);}
  printf("upper bound: %ld\n", upper);

  // allocate cpu vars
  int* const maxlen = new int;
  *maxlen = 0;

  // allocate gpu vars
  int* d_maxlen;
  if (hipSuccess != hipMalloc((void **)&d_maxlen, sizeof(int))) {fprintf(stderr, "ERROR: could not allocate memory\n"); exit(-1);}

  // initialize gpu vars
  if (hipSuccess != hipMemcpy(d_maxlen, maxlen, sizeof(int), hipMemcpyHostToDevice)) {fprintf(stderr, "ERROR: copying to device failed\n"); exit(-1);}

  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  // execute timed code
  // because we're only testing odd values, there are (upper+1)/2 number of iterations
  collatz<<<((upper+1)/2 + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(upper, d_maxlen);
  hipDeviceSynchronize();

  // end time
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0;
  printf("compute time: %.4f s\n", runtime);

  // get result from GPU
  CheckCuda();
  if (hipSuccess != hipMemcpy(maxlen, d_maxlen, sizeof(int), hipMemcpyDeviceToHost)) {fprintf(stderr, "ERROR: copying to host failed\n"); exit(-1);}

  // print result
  printf("longest sequence: %d elements\n", *maxlen);

  // clean up
  free(maxlen);
  hipFree(d_maxlen);

  return 0;
}
